
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N (4096*4096)
#define CORES (1024)

__global__ void kernel(float* dA) { 
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  float x = 2.0f * 3.1415926f * (float) idx / (float) N;
  dA[idx] = sinf(sqrtf(x));
}

bool checkLaunched() {
  hipError_t err;
  hipDeviceSynchronize();
  err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "Cannot launch CUDA kernel: %s\n", hipGetErrorString(err));
    return false;
  } 
  return true;
}

int main(void) {
  float timerValueGPU, timerValueCPU;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);
  
  float *hA, *dA;
  hA = (float*) malloc(N * sizeof(float));
  hipMalloc((void**) &dA, N * sizeof(float));
  kernel <<< N / CORES, CORES >>> (dA);
  hipMemcpy(hA, dA, N * sizeof(float), hipMemcpyDeviceToHost);

  if (!checkLaunched()) return 1;

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&timerValueGPU, start, stop);
  printf("\n GPU calculation time: %f ms\n", timerValueGPU);

  //for (int idx = 0; idx < N; idx++) {
  //  if (idx % 10000 == 0) {
  //    printf("a[%d] = %.5f\n", idx, hA[idx]);
  //  }
  //}

  hipEventRecord(start, 0);
  for (int i = 0; i < N; i++) {
    hA[i] = sinf(sqrtf(2.0f * 3.1415926f * (float) i / (float) N));
  }
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&timerValueCPU, start, stop);
  printf("\n CPU calculation time: %f ms\n", timerValueCPU);
  printf("\n Rate: %fx\n", timerValueCPU / timerValueGPU);

  free(hA); hipFree(dA);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  return 0;
}
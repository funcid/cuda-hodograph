#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include "../globals/globals.h"

__global__ void kernel(float* result, int N) 
{ 
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  result[idx] = 1.0 / (100 * (float) (idx - N / 2) / (float) N); // todo: add custom function call
}

float* internalCall() 
{
    float *hostArray, *deviceArray;
    hostArray = (float*) malloc(N * sizeof(float));
    hipMalloc((void**) &deviceArray, N * sizeof(float));
    kernel<<< N / CORES, CORES >>>(deviceArray, N);
    hipMemcpy(hostArray, deviceArray, N * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(deviceArray);
    return hostArray;
} 

float* calculate()
{
    float timerValueGPU, timerValueCPU;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    float* hostArray = internalCall();

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&timerValueGPU, start, stop);
    printf("GPU calculation time: %f ms\n", timerValueGPU);
  
    hipEventDestroy(start);
    hipEventDestroy(stop);

    float* data = new float[N * 2];
    for (int i = -N; i < N; i += 2)
    {
      data[N + i] = i * 1.0 / N;
      data[N + i + 1] = hostArray[(N + i) / 2];
    }

    delete hostArray;
    return data;
}

void checkCudaLaunched() 
{
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) 
  {
    fprintf(stderr, "Cannot launch CUDA kernel: %s\n", hipGetErrorString(err));
    exit(err);
  } 
}

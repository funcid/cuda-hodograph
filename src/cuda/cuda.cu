#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include "../globals/globals.h"

__global__ void kernel(float* result, int N) 
{ 
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  result[idx] = 1.0 / (100 * (float) (idx - N / 2) / (float) N);
}

float* call()
{
    float *hostArray, *deviceArray;
    hostArray = (float*) malloc(N * sizeof(float));
    hipMalloc((void**) &deviceArray, N * sizeof(float));
    kernel<<< N / CORES, CORES >>>(deviceArray, N);
    hipMemcpy(hostArray, deviceArray, N * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(deviceArray);
    return hostArray;
}

void checkCudaLaunched() 
{
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) 
  {
    fprintf(stderr, "Cannot launch CUDA kernel: %s\n", hipGetErrorString(err));
    exit(err);
  } 
}

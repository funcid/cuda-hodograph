#include "hip/hip_runtime.h"
#include <stdio.h>
#include <GL/glew.h>
#include <GLFW/glfw3.h>

#define WIDTH (1000)
#define HEIGHT (800)
#define N (2048*2048)
#define CORES (1024)

__global__ void kernel(float* dA) 
{ 
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  dA[idx] = 1.0 / (100 * (float) (idx - N / 2) / (float) N);
}

class VertexBufferObject
{
private:
  GLuint id;

public:
  VertexBufferObject()
  {
    glGenBuffers(1, &id);
  }

};

class VertexArrayObject
{
private:
  GLuint id;

public:
  void bind() 
  {
    glBindVertexArray(id);
  }

  void unbind() 
  {
    glBindVertexArray(0);
  }

  VertexArrayObject()
  {
    glGenVertexArrays(1, &id);
    bind();
  }
};

void checkCudaLaunched() 
{
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) 
  {
    fprintf(stderr, "Cannot launch CUDA kernel: %s\n", hipGetErrorString(err));
    exit(err);
  } 
}

void checkGlfwInit() 
{
  if (!glfwInit()) 
  {
    printf("Failed to initialize GLFW");
    exit(-389);
  } 
  else
  {
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 0);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_ANY_PROFILE);
    glfwWindowHint(GLFW_RESIZABLE, GL_TRUE);
  }
}

void framebufferSizeCallback(GLFWwindow* window, int width, int height)
{
    glViewport(0, 0, width, height);
}

void renderFrame(float* result) 
{
  glBegin(GL_LINE_STRIP);
  glVertex2f(-1, 0);
  glVertex2f(1, 0);
  glEnd();
  glBegin(GL_LINE_STRIP);
  glVertex2f(0, 1);
  glVertex2f(0, -1);
  glEnd();
  glBegin(GL_LINE_STRIP);
  for (int i = 0; i < N; i++) 
  { 
    glVertex2f(2 * (i * 1.0 / N - 0.5), result[i]);
  }
  glEnd();
}

int main(void) 
{
  checkGlfwInit();

  // CUDA calculations
  float timerValueGPU, timerValueCPU;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);
  
  float *hostArray, *deviceArray;
  hostArray = (float*) malloc(N * sizeof(float));
  hipMalloc((void**) &deviceArray, N * sizeof(float));
  kernel<<< N / CORES, CORES >>>(deviceArray);
  hipMemcpy(hostArray, deviceArray, N * sizeof(float), hipMemcpyDeviceToHost);
  checkCudaLaunched();

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&timerValueGPU, start, stop);
  printf("GPU calculation time: %f ms\n", timerValueGPU);

  hipFree(deviceArray);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  // Windows application open
  GLFWwindow* window = glfwCreateWindow(WIDTH, HEIGHT, "Hodograph", NULL, NULL);
  if (window == NULL)
  {
    printf("Failed to open GLFW window");
    exit(-341);
  }

  glfwMakeContextCurrent(window);

  glewExperimental = GL_TRUE;
  if (glewInit() != GLEW_OK) {
    printf("Failed to initialize GLEW");
    exit(-390);
  }

  glfwSetFramebufferSizeCallback(window, framebufferSizeCallback);

  printf("Renderer: %s\n", glGetString(GL_RENDERER));
  printf("OpenGL version: %s\n", glGetString(GL_VERSION));

  // GL bind array
  VertexArrayObject* object = new VertexArrayObject();

  while(!glfwWindowShouldClose(window)) 
  {
    glfwPollEvents();  
    glClear(GL_COLOR_BUFFER_BIT);
    renderFrame(hostArray);
    glfwSwapBuffers(window);
  }

  free(hostArray); 
  delete object;
  glfwTerminate();
  return 0;
}
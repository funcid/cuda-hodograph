#include "hip/hip_runtime.h"
#include <stdio.h>
#include <GL/glew.h>
#include <GLFW/glfw3.h>
#include "render/render.h"

#define WIDTH (1000)
#define HEIGHT (800)
#define N (2048*2048)
#define CORES (1024)

__global__ void kernel(float* dA) 
{ 
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  dA[idx] = 1.0 / (100 * (float) (idx - N / 2) / (float) N);
}

class VertexBufferObject
{
private:
  GLuint id;

public:
  VertexBufferObject()
  {
    glGenBuffers(1, &id);
  }

};

class VertexArrayObject
{
private:
  GLuint id;

public:
  void bind() 
  {
    glBindVertexArray(id);
  }

  void unbind() 
  {
    glBindVertexArray(0);
  }

  VertexArrayObject()
  {
    glGenVertexArrays(1, &id);
    bind();
  }
};

void checkCudaLaunched() 
{
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) 
  {
    fprintf(stderr, "Cannot launch CUDA kernel: %s\n", hipGetErrorString(err));
    exit(err);
  } 
}

void framebufferSizeCallback(GLFWwindow* window, int width, int height)
{
    glViewport(0, 0, width, height);
}

int main(void) 
{
  checkGlfwInit();

  // CUDA calculations
  float timerValueGPU, timerValueCPU;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);
  
  float *hostArray, *deviceArray;
  hostArray = (float*) malloc(N * sizeof(float));
  hipMalloc((void**) &deviceArray, N * sizeof(float));
  kernel<<< N / CORES, CORES >>>(deviceArray);
  hipMemcpy(hostArray, deviceArray, N * sizeof(float), hipMemcpyDeviceToHost);
  checkCudaLaunched();

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&timerValueGPU, start, stop);
  printf("GPU calculation time: %f ms\n", timerValueGPU);

  hipFree(deviceArray);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  // Windows application open
  GLFWwindow* window = glfwCreateWindow(WIDTH, HEIGHT, "Hodograph", NULL, NULL);
  if (window == NULL)
  {
    printf("Failed to open GLFW window");
    exit(-341);
  }

  glfwMakeContextCurrent(window);
  checkGlewInit();
  glfwSetFramebufferSizeCallback(window, framebufferSizeCallback);

  printf("Renderer: %s\n", glGetString(GL_RENDERER));
  printf("OpenGL version: %s\n", glGetString(GL_VERSION));

  // GL bind array
  VertexArrayObject* object = new VertexArrayObject();

  while(!glfwWindowShouldClose(window)) 
  {
    glfwPollEvents();  
    glClear(GL_COLOR_BUFFER_BIT);
    renderFrame(hostArray, N);
    glfwSwapBuffers(window);
  }

  free(hostArray); 
  delete object;
  glfwTerminate();
  return 0;
}
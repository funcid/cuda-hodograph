#include "hip/hip_runtime.h"
#include <stdio.h>
#include <GL/glew.h>
#include <GLFW/glfw3.h>

#define N (4096*4096)
#define CORES (1024)

__global__ void kernel(float* dA) { 
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  float x = 2.0f * 3.1415926f * (float) idx / (float) N;
  dA[idx] = sinf(sqrtf(x));
}

bool checkLaunched() {
  hipError_t err;
  hipDeviceSynchronize();
  err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "Cannot launch CUDA kernel: %s\n", hipGetErrorString(err));
    return false;
  } 
  return true;
}

void framebuffer_size_callback(GLFWwindow* window, int width, int height)
{
    //glViewport(0, 0, width, height);
}

int main(void) {
  if (!glfwInit()) {
    printf("Failed to initialize GLFW");
      return -1;
  }
  if (!glewInit()) {
    printf("Failed to initialize GLEW");
      return -1;
  }

  GLFWwindow* window;
  window = glfwCreateWindow(800, 600, "Hodograph", NULL, NULL);
  if (window == NULL)
  {
    printf("Failed to open GLFW window");
      return -1;
  }
  glfwMakeContextCurrent(window);

  printf("\nRenderer: %s", glGetString(GL_RENDERER));
  printf("\nOpenGL version: %s", glGetString(GL_VERSION));

  glViewport(0, 0, 800, 600);
  glfwSetFramebufferSizeCallback(window, framebuffer_size_callback);

  while(!glfwWindowShouldClose(window)) {
    glClear(GL_COLOR_BUFFER_BIT);

    glfwPollEvents();    
    glBegin(GL_TRIANGLES);
    glVertex2f(-0.5, -0.5);
    glVertex2f(0.0, 0.5);
    glVertex2f(0.5, -0.5);
    glEnd();
    glfwSwapBuffers(window);
  }

  glfwTerminate();

  float timerValueGPU, timerValueCPU;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);
  
  float *hA, *dA;
  hA = (float*) malloc(N * sizeof(float));
  hipMalloc((void**) &dA, N * sizeof(float));
  kernel <<< N / CORES, CORES >>> (dA);
  hipMemcpy(hA, dA, N * sizeof(float), hipMemcpyDeviceToHost);

  if (!checkLaunched()) return 1;

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&timerValueGPU, start, stop);
  printf("\n GPU calculation time: %f ms\n", timerValueGPU);

  //for (int idx = 0; idx < N; idx++) {
  //  if (idx % 10000 == 0) {
  //    printf("a[%d] = %.5f\n", idx, hA[idx]);
  //  }
  //}

  hipEventRecord(start, 0);
  for (int i = 0; i < N; i++) {
    hA[i] = sinf(sqrtf(2.0f * 3.1415926f * (float) i / (float) N));
  }
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&timerValueCPU, start, stop);
  printf("\n CPU calculation time: %f ms\n", timerValueCPU);
  printf("\n Rate: %fx\n", timerValueCPU / timerValueGPU);

  free(hA); hipFree(dA);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  return 0;
}
#include <stdio.h>
#include <GL/glew.h>
#include <GLFW/glfw3.h>
#include "render/render.h"
#include "cuda/hip/hip_runtime.h"

#define WIDTH (1000)
#define HEIGHT (800)

void framebufferSizeCallback(GLFWwindow* window, int width, int height)
{
    glViewport(0, 0, width, height);
}

int main(void) 
{
  checkGlfwInit();

  // CUDA calculations
  float* hostArray = calculate();
  checkCudaLaunched();

  // Windows application open
  GLFWwindow* window = glfwCreateWindow(WIDTH, HEIGHT, "Hodograph", NULL, NULL);
  if (window == NULL)
  {
    printf("Failed to open GLFW window");
    exit(-341);
  }

  glfwMakeContextCurrent(window);
  checkGlewInit();
  glfwSetFramebufferSizeCallback(window, framebufferSizeCallback);

  printf("Renderer: %s\n", glGetString(GL_RENDERER));
  printf("OpenGL version: %s\n", glGetString(GL_VERSION));

  while(!glfwWindowShouldClose(window)) 
  {
    glfwPollEvents();  
    glClear(GL_COLOR_BUFFER_BIT);
    renderFrame(hostArray);
    glfwSwapBuffers(window);
  }

  free(hostArray); 
  glfwTerminate();
  return 0;
}